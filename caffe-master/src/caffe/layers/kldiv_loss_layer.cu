#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/kldiv_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void KLDivLossForwardGPU(const int n, Dtype* FM, Dtype* SM, const Dtype eps, Dtype* output) {
	CUDA_KERNEL_LOOP(index, n) {
		output[index] = FM [index] * log (FM [index] / (SM [index] + eps) + eps);
	}
}

template <typename Dtype>
__global__ void KLDivLossBackwardGPU(const int n, const int map_size, const Dtype* FM, const Dtype* sum, const Dtype* SM, const Dtype eps, Dtype* output) {
	CUDA_KERNEL_LOOP(index, n) {
	    int num_idx = index / map_size;
	    output [index] = Dtype( -1) * FM [index] * (sum [num_idx] - SM [index]) /
	                       (sum [num_idx] * SM [index] + eps);
	}
}

template <typename Dtype>
void KLDivLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const int map_size_ = bottom[0]->height() * bottom[0]->width();
  const int count_ = num_ * map_size_;
  const Dtype* input_data = bottom[0]->gpu_data();
  const Dtype* target = bottom[1]->gpu_data();
  Dtype* summed_sm = summed_sm_.mutable_cpu_data();
  Dtype* normalized_fm = normalized_fm_.mutable_gpu_data();
  Dtype* tmp = normalized_fm_.mutable_gpu_diff();
  Dtype summed_fm;
  Dtype loss;
  for (int i = 0; i < num_; ++i) {
	caffe_gpu_asum(map_size_, target + i * map_size_, &summed_fm);  //summed_fm
    caffe_gpu_asum(map_size_, input_data + i * map_size_, summed_sm + i);
	caffe_gpu_scale(map_size_, Dtype(1) / (summed_fm + epsilon_), target + i * map_size_, normalized_fm + i * map_size_);  //normalized FM
	caffe_gpu_scale(map_size_, Dtype(1)/(summed_sm [i] + epsilon_), input_data + i * map_size_, tmp + i * map_size_);   //normalized SM
  }
  KLDivLossForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(count_), CAFFE_CUDA_NUM_THREADS >>>(
			count_, normalized_fm, tmp, epsilon_, tmp);
  caffe_gpu_asum(count_, tmp, &loss);  //accumulation
  top[0]->mutable_cpu_data()[0] = loss / num_;
}

template <typename Dtype>
void KLDivLossLayer<Dtype>::Backward_gpu(
    const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
    // First, compute the diff = -FM(x)*(summed_fm_-sm(X))/(summed_fm_*sm(X))
	const int map_size_ = bottom[0]->height() * bottom[0]->width();
    const int count_ = num_ * map_size_;
	const Dtype* summed_sm = summed_sm_.gpu_data();
    const Dtype* normalized_fm = normalized_fm_.gpu_data();
	const Dtype* input_data = bottom[0]->gpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
	
	KLDivLossBackwardGPU<Dtype><<<CAFFE_GET_BLOCKS(count_), CAFFE_CUDA_NUM_THREADS >>>(
			count_, map_size_, normalized_fm, summed_sm, input_data, epsilon_, bottom_diff);
	
    // Scale down gradient
    const Dtype loss_weight = top[0]->cpu_diff()[0];
    caffe_gpu_scal(count_, loss_weight / num_, bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(KLDivLossLayer);

}  // namespace caffe
