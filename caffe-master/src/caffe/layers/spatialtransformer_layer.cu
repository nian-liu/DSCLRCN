#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/proto/caffe.pb.h"
#include "caffe/layers/spatialtransformer.hpp"

namespace caffe {

template<typename Dtype>
__global__ void transform_gpu(const Dtype* input, Dtype* output, int num, int channels, 
                      int in_h, int in_w, int out_h, int out_w, bool do_mirror, 
					  SpatialTransformerParameter_Rotate rotate, bool undo) {
	if (undo) {
		if (rotate == SpatialTransformerParameter_Rotate_ROTATE90) {
			rotate = SpatialTransformerParameter_Rotate_ROTATE270;
		}
		else if (rotate == SpatialTransformerParameter_Rotate_ROTATE270) {
			rotate = SpatialTransformerParameter_Rotate_ROTATE90;
		}
	}
    CUDA_KERNEL_LOOP(index, num * channels * in_h * in_w) {
	  int n = index / (channels * in_h * in_w);
	  int c = (index % (channels * in_h * in_w)) / (in_h * in_w);
	  int h = (index % (in_h * in_w)) / in_w;
	  int w = index % in_w;
	  int top_index, bottom_index, h_idx, w_idx, temp;
	  
	  bottom_index = n * channels * in_h * in_w + c * in_h * in_w + h * in_w + w;
	  h_idx = h;
	  w_idx = w;
	  if (!undo && do_mirror) {
	   w_idx = in_w - 1 - w;
	  }
	  if (rotate == SpatialTransformerParameter_Rotate_ROTATE90) {
	   temp = w_idx;
	   w_idx = in_h - 1 - h_idx;
	   h_idx = temp;
	  } else if (rotate == SpatialTransformerParameter_Rotate_ROTATE180) {
	   w_idx = in_w - 1 - w_idx;
	   h_idx = in_h - 1 - h_idx;
	  } else if (rotate == SpatialTransformerParameter_Rotate_ROTATE270) {
	   temp = h_idx;
	   h_idx = in_w - 1 - w_idx;
	   w_idx = temp;
	  }
	  if (undo && do_mirror) {
	   w_idx = out_w - 1 - w_idx;
	  }
	  top_index = n * channels * out_h * out_w + c * out_h * out_w + h_idx * out_w + w_idx;
	  output[top_index] = input[bottom_index];
    }
}

template<typename Dtype>
void SpatialTransformerLayer< Dtype >::Forward_gpu(const vector< Blob< Dtype >* >& bottom,
                                       const vector< Blob< Dtype >* >& top) {
  const int count = bottom[0]->count();
  /* NOLINT_NEXT_LINE(whitespace/operators) */
  transform_gpu<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
           bottom[0]->gpu_data(), top[0]->mutable_gpu_data(), num_, channels_,
           height_, width_, top[0]->height(), top[0]->width(), mirror_, rotate_, 
		   operation_ == SpatialTransformerParameter_Operation_UNTRANSFORM);
  CUDA_POST_KERNEL_CHECK;
}

template<typename Dtype>
void SpatialTransformerLayer< Dtype >::Backward_gpu(const vector< Blob<Dtype>* >& top,
                                        const vector<bool>& propagate_down,
                                        const vector< Blob<Dtype>* >& bottom) {
  if (!propagate_down[0]) return;
  const int count = bottom[0]->count();
  /* NOLINT_NEXT_LINE(whitespace/operators) */
  transform_gpu<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
		   top[0]->gpu_diff(), bottom[0]->mutable_gpu_diff(), num_, channels_,
           top[0]->height(), top[0]->width(), height_, width_, mirror_, rotate_, 
            operation_ == SpatialTransformerParameter_Operation_TRANSFORM);
  CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_LAYER_GPU_FUNCS(SpatialTransformerLayer);

}  // namespace caffe