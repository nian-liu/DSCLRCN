#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/proto/caffe.pb.h"
#include "caffe/layers/srnnformat_layer.hpp"

namespace caffe {

template<typename Dtype>
__global__ void format_gpu(const Dtype* input, Dtype* output, int num, int channels, 
                      int height, int width, int N, bool gen_indicator, Dtype* indicator) {
  CUDA_KERNEL_LOOP(index, num * channels * height * width) {
	  int n = index / (channels * height * width);
	  int c = (index % (channels * height * width)) / (height * width);
	  int h = (index % (height * width)) / width;
	  int w = index % width;
	  
	  int top_index, bottom_index, T_idx = w, N_idx = n * height + h;
	  if (gen_indicator && c == 0) {
		indicator [T_idx * N + N_idx] = int (T_idx != 0);
	  }
	  bottom_index = n * channels * height * width + c * height * width + h * width + w;
	  top_index = T_idx * N * channels + N_idx * channels + c;
	  output[top_index] = input[bottom_index];
		
  }
}

template<typename Dtype>
__global__ void unformat_gpu(const Dtype* input, Dtype* output, int num, int channels, 
                      int height, int width, int N) {
  CUDA_KERNEL_LOOP(index, num * channels * height * width) {
	  int n = index / (channels * height * width);
	  int c = (index % (channels * height * width)) / (height * width);
	  int h = (index % (height * width)) / width;
	  int w = index % width;
	  
	  int top_index, bottom_index, T_idx = w, N_idx = n * height + h;
	  
	  top_index = n * channels * height * width + c * height * width + h * width + w;
	  bottom_index = T_idx * N * channels + N_idx * channels + c;
	  output[top_index] = input[bottom_index];
  }
}

template<typename Dtype>
void SRNNFormatLayer< Dtype >::Forward_gpu(const vector< Blob< Dtype >* >& bottom,
                                       const vector< Blob< Dtype >* >& top) {
  const int count = bottom[0]->count();
  if ( operation_ == SRNNFormatParameter_Operation_FORMAT ) {
    /* NOLINT_NEXT_LINE(whitespace/operators) */
    format_gpu<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
	         bottom[0]->gpu_data(), top[0]->mutable_gpu_data(), num_, channels_,
             height_, width_, N_, true, top[1]->mutable_gpu_data());
  } else {
    /* NOLINT_NEXT_LINE(whitespace/operators) */
    unformat_gpu<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
	         bottom[0]->gpu_data(), top[0]->mutable_gpu_data(), num_, channels_,
             height_, width_, N_);
  }
  CUDA_POST_KERNEL_CHECK;
}

template<typename Dtype>
void SRNNFormatLayer< Dtype >::Backward_gpu(const vector< Blob<Dtype>* >& top,
                                        const vector<bool>& propagate_down,
                                        const vector< Blob<Dtype>* >& bottom) {
  if (!propagate_down[0]) return;
  const int count = bottom[0]->count();
  if ( operation_ == SRNNFormatParameter_Operation_UNFORMAT ) {
    if (propagate_down[1])
      LOG(FATAL) << this->type()
                 << " Layer cannot backpropagate to the stream size input.";
	Dtype* nullindicator = NULL;
	/* NOLINT_NEXT_LINE(whitespace/operators) */
    format_gpu<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
	         top[0]->gpu_diff(), bottom[0]->mutable_gpu_diff(), num_, channels_,
			 height_, width_, N_, false, nullindicator);
  } else {
    /* NOLINT_NEXT_LINE(whitespace/operators) */
    unformat_gpu<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
	         top[0]->gpu_diff(), bottom[0]->mutable_gpu_diff(), num_, channels_,
             height_, width_, N_);
  }
  CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_LAYER_GPU_FUNCS(SRNNFormatLayer);

}  // namespace caffe